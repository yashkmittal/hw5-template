
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>

__global__ void findOddNumbers(const int* A, int* B, int* D, int* temp, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size){
        if (temp[tid] == 1){
            D[B[tid]] = A[tid];
        }
    }
}

__global__ void findNumOddNumbers(const int* A, int* d_temp, int* numOdd, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size){
        if (A[tid] % 2 != 0){
            atomicAdd(numOdd, 1);
            d_temp[tid] = 1;
        }
        else {
            d_temp[tid] = 0;
        }
    }
}

__global__ void upwardSweep(int* B, int stride, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size){
        if (tid % stride == 0){
            B[tid + stride - 1] += B[tid + stride / 2 - 1];
        }
    }
}

__global__ void downwardSweep(int* B, int stride, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size){
        if (tid % stride == 0){
            int leftValue = B[tid + stride / 2 - 1];
            B[tid + stride / 2 - 1] = B[tid + stride - 1];
            B[tid + stride - 1] += leftValue;
        }
    }
}



int main(int argc, char **argv)
{
    // Implement your solution for question 4. The input file is inp.txt
    // and contains an array A.
    // Running this program should output one file:
    //  (1) q4.txt which contains an array D such that D contains only the odd
    //      numbers from the input array. You should preserve the order of the
    //      numbers as they are in the input array.

    // Read input array from inp.txt
    std::ifstream inputFile("inp.txt");
    if (!inputFile){
        std::cerr << "Failed to open inp.txt" << std::endl;
        return 1;
    }

    std::vector<int> A;
    std::string line;
    while (std::getline(inputFile, line, ',')){
        int value = std::stoi(line);
        A.push_back(value);
    }
    inputFile.close();
    int size = A.size();

    // Find the smallest power of 2 that is greater than or equal to the size of A
    int powerOfTwo = 1;
    int logSize = 0;
    while (powerOfTwo < size) {
        powerOfTwo *= 2;
        logSize++;
    }
    // Expand A with 0 elements to make its length a power of 2
    A.resize(powerOfTwo, 0);
    size = A.size();

    int* d_A;
    int* d_temp;
    int* d_numOdd;
    int numOdd = 0;
    
    // Allocate memory on the GPU
    hipMalloc((void**)&d_A, size * sizeof(int));
    hipMalloc((void**)&d_temp, size * sizeof(int));
    hipMalloc((void**)&d_numOdd, sizeof(int));

    // Copy input array from host to device
    hipMemcpy(d_A, A.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numOdd, &numOdd, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to find number of odd numbers
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    findNumOddNumbers<<<gridSize, blockSize>>>(d_A, d_temp, d_numOdd, size);

    // Copy number of odd numbers from device to host
    hipMemcpy(&numOdd, d_numOdd, sizeof(int), hipMemcpyDeviceToHost);

    // Blelloch Scan
    int* d_B;
    hipMalloc((void**)&d_B, size * sizeof(int));
    hipMemcpy(d_B, d_temp, size * sizeof(int), hipMemcpyDeviceToDevice);

    for(int h = 0; h < logSize; h++){
        int stride = 1 << (h + 1);
        upwardSweep<<<gridSize, blockSize>>>(d_B, stride, size);
    }
    int zero = 0;
    hipMemcpy(&d_B[size - 1], &zero, sizeof(int), hipMemcpyHostToDevice);
    for(int h = logSize - 1; h >= 0; h--){
        int stride = 1 << (h + 1);
        downwardSweep<<<gridSize, blockSize>>>(d_B, stride, size);
    }
 
    // Allocate memory for array D on the GPU
    int* d_D;
    hipMalloc((void**)&d_D, numOdd * sizeof(int));

    // Launch kernel to find and copy odd numbers to array D
    findOddNumbers<<<gridSize, blockSize>>>(d_A, d_B, d_D, d_temp, size);

    // Copy array D from device to host
    std::vector<int> D(numOdd);
    hipMemcpy(D.data(), d_D, numOdd * sizeof(int), hipMemcpyDeviceToHost);

    // Write array D to q4.txt
    std::ofstream outputFile("q4.txt");
    if (!outputFile){
        std::cerr << "Failed to open q4.txt" << std::endl;
        return 1;
    }

    for (int i = 0; i < numOdd; i++){
        if(i == numOdd - 1){
            outputFile << D[i];
        }
        else{
            outputFile << D[i] << ", ";
        }
    }
    outputFile.close();

    // Free memory
    hipFree(d_A);
    hipFree(d_temp);
    hipFree(d_numOdd);
    hipFree(d_D);

    return 0;
}
